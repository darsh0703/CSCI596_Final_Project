#include "hip/hip_runtime.h"
#include "simulation.h"

#include <hip/hip_runtime.h>
#include <>

#include "config.h"

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include <cmath>
#include <glm/geometric.hpp>
#include <glm/gtc/constants.hpp>

#include "cuda_common.h"
#include "radix_sort.h"

struct SimulationData
{
	float3 * positions;
	float3 * velocities;
	float * densities;
	float * pressures;
	float3 * accelerations;
};

// CUDA Constants
__constant__ __device__ struct SimulationConfig dSimCfg;
hiprandState_t * gRngStates;

// CUDA Kernel Function Forward Declarations
__global__ void cuInitializeParticles(SimulationData sim, hiprandState_t * rngStates, size_t count);
__global__ void cuSimulateParticles(SimulationData sim, hiprandState_t * rngStates, size_t count, float deltaTime, float totalTime);

thread_local static hipError_t cuError;

// Host calling functions
Result Simulation::__initializeParticles()
{
	cudaCall(hipMemcpyToSymbol, dSimCfg, &gSimCfg, sizeof(SimulationConfig));

	cudaCall(hipMalloc, &gRngStates, sizeof(hiprandState_t) * gSimCfg.NumParticles);

	cudaCall(hipMalloc, &cellStarts, sizeof(uint32_t) * gSimCfg.NumCells);
	cudaCall(hipMalloc, &particleCellIdx, sizeof(uint32_t) * gSimCfg.NumParticles);

#ifdef FLSIM_CHECK_DEV_SIMCFG
	SimulationConfig tmpCfg;
	hipMemcpyFromSymbol(&tmpCfg, HIP_SYMBOL(dSimCfg), sizeof(SimulationConfig));
	printf("NumParticles: %zu\n", tmpCfg.NumParticles);
	printf("ThreadGridDim: (%u, %u, %u)\n", tmpCfg.ThreadGridDim.x, tmpCfg.ThreadGridDim.y, tmpCfg.ThreadGridDim.z);
	printf("ThreadBlockDim: (%u, %u, %u)\n", tmpCfg.ThreadBlockDim.x, tmpCfg.ThreadBlockDim.y, tmpCfg.ThreadBlockDim.z);
	printf("Region: (%f, %f, %f)\n", tmpCfg.Region.x, tmpCfg.Region.y, tmpCfg.Region.z);
#endif

	velocities    = (float3*)&positions[gSimCfg.NumParticles];
	densities     = (float*)&velocities[gSimCfg.NumParticles];
	pressures     = (float*)&densities[gSimCfg.NumParticles];
	accelerations = (float3*)&pressures[gSimCfg.NumParticles];

	SimulationData inputs { positions, velocities, densities, pressures, accelerations };
	cudaKernelCall(cuInitializeParticles, gSimCfg.ThreadGridDim, gSimCfg.ThreadBlockDim, inputs, gRngStates, gSimCfg.NumParticles);

	return FLSIM_SUCCESS;
}

Result Simulation::__initializeCells()
{

	return FLSIM_SUCCESS;
}

Result Simulation::__simulateParticles(float deltaTime, float totalTime)
{
	SimulationData inputs { positions, velocities, densities, pressures, accelerations };
	cudaKernelCall(cuSimulateParticles, gSimCfg.ThreadGridDim, gSimCfg.ThreadBlockDim, inputs, gRngStates, gSimCfg.NumParticles, deltaTime, totalTime);

	return FLSIM_SUCCESS;
}

Result Simulation::__updateCells()
{
	return FLSIM_SUCCESS;
}


/***************************************************************
 * CUDA Kernel Function Definitions
 **************************************************************/

__device__ uint32_t cuGlobalIndex()
{
	uint32_t threadNumInBlock = threadIdx.x * (blockDim.y * blockDim.z) + threadIdx.y * (blockDim.z) + threadIdx.z;
    uint32_t blockNumInGrid = blockIdx.x * (gridDim.y * gridDim.z) + blockIdx.y * (gridDim.z) + blockIdx.z;
    uint32_t threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;

	return threadNumInBlock + blockNumInGrid * threadsPerBlock;
}

__device__ dim3 cuVectorIndex(int gid)
{
	dim3 NThreads = dim3(gridDim.x * blockDim.x, gridDim.y * blockDim.y, gridDim.z * blockDim.z);

	uint32_t xidx = gid / (NThreads.y * NThreads.z);
	uint32_t yidx = (gid / NThreads.z) % NThreads.y;
	uint32_t zidx = gid % NThreads.z;

	/*uint32_t xidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t yidx = threadIdx.y + blockIdx.y * blockDim.y;
    uint32_t zidx = threadIdx.z + blockIdx.z * blockDim.z;*/
    return { xidx, yidx, zidx };
}

__device__ dim3 cuCellVectorIndex(float3 position)
{
	float3 cellSize = dSimCfg.CellSize;
	float3 regionHalf = dSimCfg.RegionHalf;

	uint32_t xidx = (position.x + regionHalf.x) / cellSize.x;
	uint32_t yidx = (position.y + regionHalf.y) / cellSize.y;
	uint32_t zidx = (position.z + regionHalf.z) / cellSize.z;

	return { xidx, yidx, zidx };
}

__device__ uint32_t cuCellIndex(dim3 vid)
{
	return vid.x * dSimCfg.CellGridDim.y * dSimCfg.CellGridDim.z + vid.y * dSimCfg.CellGridDim.z + vid.z;
}

__device__ void cuOscillate(float& val, float deltaTime, float totalTime)
{
	float frequency = 2.0f;
	float amplitude = 2.4f;
	val += amplitude * std::cos(frequency * totalTime) * deltaTime;
}

__device__ glm::vec3 cuRandomDirection(hiprandState_t * rngState)
{
	float theta = hiprand_uniform(rngState) * glm::two_pi<float>();
	float phi   = hiprand_uniform(rngState) * glm::pi<float>();

	glm::vec3 randVec;
	randVec.x = sin(phi) * cos(theta);
	randVec.y = sin(phi) * sin(theta);
	randVec.z = cos(phi);

	return randVec;
}

__device__ float cuWendlandC2(float h, float r)
{
	float alpha = 21.f / (16.f * glm::pi<float>() * h * h * h);
	float WC2 = 0.f;
	float q = r / h;

	if (q < 2.f) {
		float term1 = 1 - q / 2.f;
		term1 = term1 * term1;
		term1 = term1 * term1;

		float term2 = 2 * q + 1;

		WC2 = alpha * term1 * term2;
	}

	return WC2;
}

__device__ float cuWendlandC2Derivative(float h, float r)
{
	float alpha = -105.f / (16.f * glm::pi<float>() * h * h * h);
	float WC2 = 0.f;
	float q = r / h;

	if (q < 2.f) {
		if (r > 1e-8f) {
			float term1 = 1 - q / 2.f;
			term1 = term1 * term1 * term1;

			WC2 = alpha * term1 * q;
		}
	}

	return WC2;
}

__device__ void cuResolveCollisions(glm::vec3& position, glm::vec3& velocity)
{
	if (fabsf(position.x) > dSimCfg.RegionHalf.x) {
		position.x = copysignf(dSimCfg.RegionHalf.x, position.x);
		velocity.x *= -1 * dSimCfg.CollisionDamping;
	}
	if (fabsf(position.y) > dSimCfg.RegionHalf.y) {
		position.y = copysignf(dSimCfg.RegionHalf.y, position.y);
		velocity.y *= -1 * dSimCfg.CollisionDamping;
	}
	if (fabsf(position.z) > dSimCfg.RegionHalf.z) {
		position.z = copysignf(dSimCfg.RegionHalf.z, position.z);
		velocity.z *= -1 * dSimCfg.CollisionDamping;
	}
}

__device__ float cuCalculateDensity(float3 * positions, size_t count, uint32_t gid)
{
	float density = 0.f;

	glm::vec3 position = *(glm::vec3*)&positions[gid];

	for (uint32_t i = 0; i < count; i++) {
		float dist = glm::distance(*(glm::vec3*)&positions[i], position);

		if (dist < 2 * dSimCfg.SmoothingRadius) {
			float W = cuWendlandC2(dSimCfg.SmoothingRadius, dist);
			density += dSimCfg.ParticleMass * W;
		}
	}

	return density;
}

__device__ float cuCalculatePressure(float density)
{
	float pressure = dSimCfg.ReferencePressure * (powf(density / dSimCfg.ReferenceDensity, dSimCfg.TaitExponent) - 1);
	//return __max(pressure, 0.00001f);
	return pressure;
}

__device__ glm::vec3 cuCalculatePressureAcceleration(SimulationData sim, hiprandState_t * rngStates, size_t count, uint32_t gid)
{
	glm::vec3 pressureAcc { 0.f };

	glm::vec3 position = *(glm::vec3*)&sim.positions[gid];
	glm::vec3 velocity = *(glm::vec3*)&sim.velocities[gid];
	float     density  = sim.densities[gid];
	float     pressure = sim.pressures[gid];
	float     density2 = density * density;

	density += glm::epsilon<float>();
	density2 += glm::epsilon<float>();

	for (uint32_t i = 0; i < count; i++) {
		if (i == gid) continue;
		glm::vec3 deltaPos = *(glm::vec3*)&sim.positions[i] - position;
		float dist = glm::length(deltaPos);

		if (dist < 2 * dSimCfg.SmoothingRadius) {
			float dW = cuWendlandC2Derivative(dSimCfg.SmoothingRadius, dist);

			glm::vec3 dir = dist < 1e-8f ? cuRandomDirection(&rngStates[gid]) : deltaPos / dist;
			glm::vec3 delW = dir * dW;

			pressureAcc += density * dSimCfg.ParticleMass * ((pressure / density2) + (sim.pressures[i] / (sim.densities[i] * sim.densities[i]))) * delW;
			// pressureAcc += - dSimCfg.ParticleMass * sim.pressures[i] / sim.densities[i] * delW;
			// pressureAcc += dSimCfg.ParticleMass / density * (pressure + sim.pressures[i]) * delW;
		}
	}
	pressureAcc *= 1 / density;

	return pressureAcc;
}

__device__ glm::vec3 cuCalculateViscosityAcceleration(SimulationData sim, hiprandState_t * rngStates, size_t count, uint32_t gid)
{
	glm::vec3 viscosityAcc { 0.f };

	glm::vec3 position = *(glm::vec3*)&sim.positions[gid];
	glm::vec3 velocity = *(glm::vec3*)&sim.velocities[gid];
	float     density  = sim.densities[gid];
	float     pressure = sim.pressures[gid];
	float     density2 = density * density;

	density += glm::epsilon<float>();
	density2 += glm::epsilon<float>();

	for (uint32_t i = 0; i < count; i++) {
		if (i == gid) continue;
		glm::vec3 deltaPos = *(glm::vec3*)&sim.positions[i] - position;
		glm::vec3 deltaVel = *(glm::vec3*)&sim.velocities[i] - velocity;
		float dist = glm::length(deltaPos);
		dist += glm::epsilon<float>();

		if (dist < 2 * dSimCfg.SmoothingRadius) {
			float dW = cuWendlandC2Derivative(dSimCfg.SmoothingRadius, dist);

			glm::vec3 dir = dist < 1e-8f ? cuRandomDirection(&rngStates[gid]) : deltaPos / dist;
			glm::vec3 delW = dir * dW;
			float normDelW = glm::length(delW);

			viscosityAcc += dSimCfg.ParticleMass / density * dSimCfg.Viscosity * 2.f * deltaVel * normDelW / dist;
		}
	}

	return viscosityAcc;
}

__global__ void cuInitializeParticles(SimulationData sim, hiprandState_t * rngStates, size_t count)
{
	dim3 globalDim = dim3(gridDim.x * blockDim.x, gridDim.y * blockDim.y, gridDim.z * blockDim.z);
	size_t nThreads = globalDim.x * globalDim.y * globalDim.z;
	uint32_t gid = cuGlobalIndex();
	dim3 vid = cuVectorIndex(gid);

	hiprand_init(53135, gid, 0, &rngStates[gid]);

	float3 noise      = { (hiprand_uniform(&rngStates[gid]) - 0.5f) / nThreads, (hiprand_uniform(&rngStates[gid]) - 0.5f) / nThreads, (hiprand_uniform(&rngStates[gid]) - 0.5f) / nThreads };
	float3 fraction   = { (float)vid.x / (float)globalDim.x, (float)vid.y / (float)globalDim.y, (float)vid.z / (float)globalDim.z };
	float3 distance   = { noise.x + fraction.x * dSimCfg.InitRegion.x, noise.y + fraction.y * dSimCfg.InitRegion.y, noise.z + fraction.z * dSimCfg.InitRegion.z };
	float3 position   = { distance.x + dSimCfg.InitPosition.x, distance.y + dSimCfg.InitPosition.y, distance.z + dSimCfg.InitPosition.z };

	if (gid < count) {
		sim.positions[gid] = { position.x, position.y, position.z };
		sim.velocities[gid] = { 0.f, 0.f, 0.f };

		__syncthreads();

		float density = cuCalculateDensity(sim.positions, count, gid);
		sim.densities[gid] = density;
		__syncthreads();

		sim.pressures[gid] = cuCalculatePressure(sim.densities[gid]);
		__syncthreads();

		glm::vec3 pressureAcc = cuCalculatePressureAcceleration(sim, rngStates, count, gid);
		glm::vec3 viscosityAcc = cuCalculateViscosityAcceleration(sim, rngStates, count, gid);

		glm::vec3 acceleration = pressureAcc + viscosityAcc + (*(glm::vec3*)&dSimCfg.Gravity);

		sim.accelerations[gid] = *(float3*)&acceleration;
	}
}

__global__ void cuSimulateParticles(SimulationData sim, hiprandState_t * rngStates, size_t count, float deltaTime, float totalTime)
{
	uint32_t gid = cuGlobalIndex();

	if (gid < count) {
		glm::vec3 prev_position     = *(glm::vec3*)&sim.positions[gid];
		glm::vec3 prev_velocity     = *(glm::vec3*)&sim.velocities[gid];

		glm::vec3 position     = prev_position;
		glm::vec3 velocity     = prev_velocity;
		float     density;
		float     pressure;

		glm::vec3 pressureAcc = cuCalculatePressureAcceleration(sim, rngStates, count, gid);
		glm::vec3 viscosityAcc = cuCalculateViscosityAcceleration(sim, rngStates, count, gid);
		glm::vec3 acceleration = pressureAcc + viscosityAcc + (*(glm::vec3*)&dSimCfg.Gravity);

		// half tap
		velocity += 0.5f * deltaTime * acceleration;
		position += 0.5f * deltaTime * velocity;
		cuResolveCollisions(position, velocity);
		density  = cuCalculateDensity(sim.positions, count, gid);
		pressure = cuCalculatePressure(density);

		sim.velocities[gid] = *(float3*)&velocity;
		sim.positions[gid]  = *(float3*)&position;
		sim.densities[gid]  = density;
		sim.pressures[gid]  = pressure;
		__syncthreads();

		pressureAcc = cuCalculatePressureAcceleration(sim, rngStates, count, gid);
		viscosityAcc = cuCalculateViscosityAcceleration(sim, rngStates, count, gid);
		acceleration = pressureAcc + viscosityAcc + (*(glm::vec3*)&dSimCfg.Gravity);

		// full tap
		velocity = prev_velocity + deltaTime * acceleration;
		position = prev_position + deltaTime * velocity;
		cuResolveCollisions(position, velocity);
		density  = cuCalculateDensity(sim.positions, count, gid);
		pressure = cuCalculatePressure(density);


		sim.velocities[gid]    = *(float3*)&velocity;
		sim.positions[gid]     = *(float3*)&position;
		sim.densities[gid]     = density;
		sim.pressures[gid]     = pressure;
		sim.accelerations[gid] = *(float3*)&acceleration; // for plotting only
	}
}




//for (uint32_t i = 0; i < count; i++) {
		//	if (i == gid) continue;
		//	glm::vec3 deltaPos = *(glm::vec3*)&positions[i] - position;
		//	float dist = glm::length(deltaPos);

		//	if (dist < 2 * dSimCfg.SmoothingRadius) {
		//		glm::vec3 deltaVel = *(glm::vec3*)&velocities[i] - velocity;

		//		float W = cuWendlandC2(dSimCfg.SmoothingRadius, dist);
		//		glm::vec3 delWx { cuWendlandC2(dSimCfg.SmoothingRadius, deltaPos.x), cuWendlandC2(dSimCfg.SmoothingRadius, deltaPos.y), cuWendlandC2(dSimCfg.SmoothingRadius, deltaPos.z) };

		//		newDensity += dSimCfg.ParticleMass * W;
		//		divVel  += dSimCfg.ParticleMass * glm::dot(deltaVel, delWx);

		//		pressureGradTerm += - dSimCfg.ParticleMass * ((pressure / density2) + (pressures[i] / (densities[i] * densities[i]))) * delWx;
		//		//viscosityTerm    += dSimCfg.Viscosity / pressure * dSimCfg.ParticleMass * 4 / ((pressure + pressures[i]) * glm::dot(deltaPos, deltaPos)) * (glm::dot(deltaPos, delWx)) * deltaVel;
		//	}
		//}